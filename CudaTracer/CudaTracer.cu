#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <gl/glew.h>
#include <gl/GL.h>
#include <gl/freeglut.h>
#include "cuda_gl_interop.h"
#include "CudaUtils.h"
#include <math.h>
#include "Primitives.h"
#include "CudaTracer.h"
#pragma comment(lib, "glew32.lib")

//TODO: These ought to be done differently

__global__ void kernel(uchar4 *pixels, RendererData data)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	//create the ray
	Ray tempRay = computeEyeRay(x, y, DIM, DIM, data.camera);

	//loop through spheres, finding intersection
	float distance = INFINITY;
	SurfaceElement surfel;
	char intersection = 0;
	short3 radiance;
	radiance.x = 0;
	radiance.y = 0;
	radiance.z = 0;

	//TODO: Refactor into reusable way to find an intersection
	for (size_t i = 0; i < data.numSpheres; i++)
	{
		Sphere sphere = data.spheres[i];

		if (sphere.intersectRay(tempRay, distance, surfel))
		{
			intersection = 1;
		}
	}

	for (size_t i = 0; i < data.numTriangles; i++)
	{
		Triangle triangle = data.triangles[i];

		if (triangle.intersectRay(tempRay, distance, surfel))
		{
			intersection = 1;
		}
	}

	if (intersection)
	{	
		//TODO
		Material material = data.materials[surfel.materialIdx];

		//intersection found, calculate direct light
		radiance = shade(data, surfel, material, tempRay.direction);
	}
	else
	{
		radiance.x = data.defaultColor.x;
		radiance.y = data.defaultColor.y;
		radiance.z = data.defaultColor.z;
		tempRay.alive = 0;
	}

	//save the ray
	data.rays[offset] = tempRay;

	//access uchar4
	pixels[offset].x = static_cast<unsigned char>(glm::clamp<short>(radiance.x, 0, 255));
	pixels[offset].y = static_cast<unsigned char>(glm::clamp<short>(radiance.y, 0, 255));
	pixels[offset].z = static_cast<unsigned char>(glm::clamp<short>(radiance.z, 0, 255));
	pixels[offset].w = 255;
}

__device__ 
Ray computeEyeRay(int x, int y, int dimX, int dimY, Camera* camera)
{
	const float aspectRatio = float(dimY) / dimX;

	// Compute the side of a square at z = -1 (the far clipping plane) based on the 
	// horizontal left-edge-to-right-edge field of view

	//multiplying by negative 2 offsets the -.5 in the next step
	const float s = -2 * tan(camera->fieldOfView * 0.5f);

	// xPos / image.width() : map from 0 - 1 where the pixel is on the image
	const vec3 start = vec3(((float(x) / dimX) - 0.5f) * s,
		1 * ((float(y) / dimY) - 0.5f) * s * aspectRatio,
		1.0f)
		* camera->zNear;

	return Ray(camera->position, glm::normalize(camera->rotation * start));
}

__device__
short3 shade(const RendererData& data, const SurfaceElement& surfel, const Material& material, const vec3& w_o)
{
	vec3 w_i;
	float distance2;
	short3 radiance;		//result may surpass 255, so an int3 is used
	radiance.x = 0;
	radiance.y = 0;
	radiance.z = 0;


	//TODO: loop through all lights
	for (size_t i = 0; i < data.numPointLights; i++)
	{
		PointLight light = data.pointLights[i];

		if (lineOfSight(data, surfel.point, light.position, w_i, distance2))
		{
			const vec3 L_i = light.power / float(4 * M_PI * distance2);

			float cosI = fmax(0.0f, dot(surfel.normal, w_i));

			radiance.x += 255 * (cosI * L_i.r * material.diffuseColor.r * material.diffAvg / M_PI);
			radiance.y += 255 * (cosI * L_i.g * material.diffuseColor.g * material.diffAvg / M_PI);
			radiance.z += 255 * (cosI * L_i.b * material.diffuseColor.b * material.diffAvg / M_PI);
		}
	}

	return radiance;
}

__device__
bool lineOfSight(const RendererData& data, const vec3& point0, const vec3& point1, vec3& w_i, float& distance2)
{
	const vec3 offset = point1 - point0;
	distance2 = dot(offset, offset);
	float distance = sqrt(distance2);

	w_i = offset / distance;

	const Ray losRay(point0 + (RAY_BUMP_EPSILON * w_i), w_i);

	//shorten distance.
	distance -= RAY_BUMP_EPSILON;

	//loop through all primitives, seeing if any intersections occur
	SurfaceElement surfel;

	//TODO: More robust implementation
	for (size_t i = 0; i < data.numSpheres; i++)
	{
		Sphere sphere = data.spheres[i];

		if (sphere.intersectRay(losRay, distance, surfel))
		{
			return false;
		}
	}

	for (size_t i = 0; i < data.numTriangles; i++)
	{
		Triangle triangle = data.triangles[i];

		if (triangle.intersectRay(losRay, distance, surfel))
		{
			return false;
		}
	}

	return true;
}

void generateFrame(uchar4 *pixels, void* dataBlock, int ticks)
{
	RendererData *data = (RendererData *)dataBlock;
	

	dim3 grids(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	kernel <<< grids, threads >>>(pixels, *data);
}

int main(int argc, char *argv[])
{
	Scene scene;

	uchar3 defaultColor;
	defaultColor.x = 100;
	defaultColor.y = 100;
	defaultColor.z = 100;

	buildScene(scene);

	Camera *camera;
	PointLight *pointLights;
	AreaLight *areaLights;
	Ray* rays;
	Sphere *spheres;
	Triangle *triangles;
	Material *materials;

	//initialize bitmap and data
	RendererData *data = new RendererData();
	GPUAnimBitmap bitmap(DIM, DIM, data);

	//allocate GPU pointers
	CUDA_ERROR_HANDLE(hipMalloc((void**)&camera,
		sizeof(Camera)));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&pointLights,
		sizeof(PointLight)* scene.pointLightsVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&areaLights,
		sizeof(AreaLight)* scene.areaLightsVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&spheres,
		sizeof(Sphere)* scene.spheresVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&materials,
		sizeof(Material)* scene.materialsVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&triangles,
		sizeof(Triangle)* scene.trianglesVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&rays,
		sizeof(Ray)* DIM * DIM));
	
	//initialize values
	Camera temp_c = Camera();

	//copy data to GPU
	CUDA_ERROR_HANDLE(hipMemcpy(camera, &temp_c, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(spheres, scene.spheresVec.data(), sizeof(Sphere)* scene.spheresVec.size(), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(triangles, scene.trianglesVec.data(), sizeof(Triangle)* scene.trianglesVec.size(), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(pointLights, scene.pointLightsVec.data(), sizeof(PointLight)* scene.pointLightsVec.size(), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(areaLights, scene.areaLightsVec.data(), sizeof(AreaLight)* scene.areaLightsVec.size(), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(materials, scene.materialsVec.data(), sizeof(Material)* scene.materialsVec.size(), hipMemcpyHostToDevice));

	//put values in a data block
	data->camera = camera;
	data->pointLights = pointLights;
	data->numPointLights = scene.pointLightsVec.size();
	data->areaLights = areaLights;
	data->numAreaLights = scene.areaLightsVec.size();
	data->spheres = spheres;
	data->numSpheres = scene.spheresVec.size();
	data->triangles = triangles;
	data->numTriangles = scene.trianglesVec.size();
	data->rays = rays;
	data->materials = materials;
	data->defaultColor = defaultColor;



	bitmap.anim_and_exit((void(*)(uchar4*, void*, int))generateFrame, NULL, (void(*)(unsigned char, int, int))Key);

	//free
	CUDA_ERROR_HANDLE(hipFree(camera));
	CUDA_ERROR_HANDLE(hipFree(pointLights));
	CUDA_ERROR_HANDLE(hipFree(areaLights));
	CUDA_ERROR_HANDLE(hipFree(spheres));
	CUDA_ERROR_HANDLE(hipFree(triangles));
	CUDA_ERROR_HANDLE(hipFree(rays));
	CUDA_ERROR_HANDLE(hipFree(materials));
	
	delete data;

	return 0;
}

void buildScene(Scene& scene) {
	float power = 800;

	//add point light
	scene.pointLightsVec.push_back(PointLight(vec3(-2, 4.0f, 0), vec3(power, power, power)));
//	scene.pointLightsVec.push_back(PointLight(vec3(2, 1.0f, 0), vec3(power, power, power)));

	//add Spheres
	addRandomSpheres(scene, 20);

	//add cornell box
	addCornellBox(scene, 10);
}

void addRandomSpheres(Scene& scene, const size_t numSpheres)
{
	int matIdx = scene.materialsVec.size();

	//add matrials
	scene.materialsVec.push_back(Material(vec3(0, 1.0f, 1.0f), 0.9f));
	scene.materialsVec.push_back(Material(vec3(1.0f, 1.0f, 1.0f), 0.9f));
	scene.materialsVec.push_back(Material(vec3(1.0f, 0.0f, 0.0f), 0.9f));

	for (int i = 0; i < numSpheres; i++)
	{
		Sphere s;

		s.position = vec3(rnd(6.0f) - 3.0f, rnd(6.0f) - 3.0f, rnd(5.0f) - 8.0f);
		s.radius = rnd(0.5f) + 0.5f;
		s.materialIdx = matIdx + (i % 3);

		scene.spheresVec.push_back(s);
	}
}

void addCornellBox(Scene& scene, const float wallSize)
{
	using glm::translate;
	using glm::scale;
	using glm::rotate;
	
	int matIdx = scene.materialsVec.size();

	scene.materialsVec.push_back(Material(vec3(1.0f, 1.0f, 0.8f), 0.9f));	//white			(+0)
	scene.materialsVec.push_back(Material(vec3(1.0f, 0.0f, 0.0f), 0.9f));	//red			(+1)
	scene.materialsVec.push_back(Material(vec3(0.0f, 1.0f, 0.0f), 0.9f));	//green			(+2)
	scene.materialsVec.push_back(Material(vec3(1.0f, 1.0f, 1.0f)));			//white light	(+3)

	const float offset = wallSize / 2;

	const mat4 scaleToWall = scale(vec3(wallSize, wallSize, wallSize));

	//floor
	mat4 trans = translate(vec3(0, -offset, -offset)) *
		rotate(-(glm::mediump_float)90, vec3(1, 0, 0)) *
		scaleToWall;
	scene.addRectangularModel(trans, matIdx);

	//ceiling
	trans = translate(vec3(0, offset, -offset)) *
		rotate((glm::mediump_float)90, vec3(1, 0, 0)) *
		scaleToWall;
	scene.addRectangularModel(trans, matIdx);

	//left wall
	trans = translate(vec3(-offset, 0, -offset)) *
		rotate((glm::mediump_float)90, vec3(0, 1, 0)) *
		scaleToWall;
	scene.addRectangularModel(trans, matIdx + 1);

	//right wall
	trans = translate(vec3(offset, 0, -offset)) *
		rotate((glm::mediump_float)90, vec3(0, 1, 0)) *
		scaleToWall;
	scene.addRectangularModel(trans, matIdx + 2);

	//back wall
	trans = translate(vec3(0, 0, -wallSize)) *
//		rotate((glm::mediump_float)90, vec3(1, 0, 0)) *
		scaleToWall;
	scene.addRectangularModel(trans, matIdx);
}

//TODO: Clean - doesn't need to be a kernel
__global__ void moveCamera(Camera *camera, unsigned char key)
{
	switch (key) {
		case 119:
		{
					//forward (w)
					camera->position.z -= .1f;
					break;
		}
		case 97:
		{
				   //left (a)
				   camera->position.x -= .1f;
				   break;
		}
		case 115:
		{
					//backwards (s)
					camera->position.z += .1f;
					break;
		}
		case 100:
		{
					//right (d)
					camera->position.x += .1f;
					break;
		}
		case 113:
		{
				   //up (q)
				   camera->position.y += .1f;
				   break;
		}
		case 101:
		{
				   //down (e)
				   camera->position.y -= .1f;
				   break;
		}

	}
}


// static method used for glut callbacks
void Key(unsigned char key, int x, int y) {

	GPUAnimBitmap*   bitmap = *(GPUAnimBitmap::get_bitmap_ptr());

	switch (key) {
		case 27:
		{
			   if (bitmap->animExit)
				   bitmap->animExit(bitmap->dataBlock);
			   bitmap->free_resources();
			   exit(0);
		}
	}

	moveCamera <<< 1, 1 >>>(((RendererData*)bitmap->dataBlock)->camera, key);
}
