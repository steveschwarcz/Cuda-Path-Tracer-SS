#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <fstream>
#include <time.h>
#include <cmath>
#include <gl/glew.h>
#include <gl/GL.h>
#include <gl/freeglut.h>
#include "cuda_gl_interop.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include "CudaUtils.h"
#include "Primitives.h"
#include "CudaTracer.h"
#pragma comment(lib, "glew32.lib")

//TODO: These ought to be done differently

//Initialize hiprand states
__global__ void curandSetupKernel(hiprandState *state)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	/* Each thread gets same seed, a different sequence number, no offset */ 
	hiprand_init((unsigned int)clock64(), offset, 0, &state[offset]);
}

__global__ void clearPixels(uchar4 *pixels, uint3 *totalPixelColors) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x; 
	
	uchar4 newPixel;
	newPixel.x = 0;
	newPixel.y = 0;
	newPixel.z = 0;
	newPixel.w = 0;

	uint3 newPixelTotal;
	newPixelTotal.x = 0;
	newPixelTotal.y = 0;
	newPixelTotal.z = 0;

	pixels[offset] = newPixel;
	totalPixelColors[offset] = newPixelTotal;
}

__global__ void computeEyeRaysKernel(Camera camera, Ray* rays, hiprandState* states) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	Ray ray = computeEyeRay(x, y, DIM, DIM, camera, states[offset]);

	ray.pixelOffset = offset;

	rays[offset] = ray;
}

__global__ void writeToPixelsKernel(uchar4 *pixels, uint3 *totalPixelColors, Ray* rays, int ticks) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	Ray ray = rays[offset];

	int pixelOffset = ray.pixelOffset;

	vec3 radiance = glm::clamp(ray.radiance0, 0.0f, 1.0f);

	//Gamma correction
	radiance.x = pow(radiance.x, GAMMA_CORRECTION);
	radiance.y = pow(radiance.y, GAMMA_CORRECTION);
	radiance.z = pow(radiance.z, GAMMA_CORRECTION);

	uchar3 newPixel;
	uint3 totalPixels = totalPixelColors[pixelOffset];		//the average of the current pixel, multiplied by number of samples (ticks)

	//update pixel
	newPixel.x = static_cast<unsigned int>(glm::clamp<float>(255 * radiance.x + 0.5f, 0.f, 255.f));
	newPixel.y = static_cast<unsigned int>(glm::clamp<float>(255 * radiance.y + 0.5f, 0.f, 255.f));
	newPixel.z = static_cast<unsigned int>(glm::clamp<float>(255 * radiance.z + 0.5f, 0.f, 255.f));

	//now average the pixels
	uchar4 currentPixel;

	totalPixels.x += newPixel.x;
	totalPixels.y += newPixel.y;
	totalPixels.z += newPixel.z;

	float inverseTicks = 1.f / (ticks + 1);
	currentPixel.x = static_cast<unsigned char>(totalPixels.x * inverseTicks + 0.5f);
	currentPixel.y = static_cast<unsigned char>(totalPixels.y * inverseTicks + 0.5f);
	currentPixel.z = static_cast<unsigned char>(totalPixels.z * inverseTicks + 0.5f);
	currentPixel.w = 255;

	totalPixelColors[pixelOffset] = totalPixels;
	pixels[pixelOffset] = currentPixel;

	return;
}

__global__ void pathTraceKernel(uchar4 *pixels, RendererData data, int iterations)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;

	//get hiprand state
	hiprandState localState = data.curandStates[offset];

	Ray ray(true);

	//use the existing ray
	ray = data.rays[offset];

	//before continuing, be sure that the ray is active
	if (!ray.active) {
		return;
	}

	float distance = INFINITY;
	SurfaceElement surfel;
	char intersection = 0;

	//TODO: Refactor into reusable way to find an intersection
	//loop through spheres, finding intersection
	for (size_t i = 0; i < data.numSpheres; i++)
	{
		Sphere sphere = data.spheres[i];

		if (sphere.intersectRay(ray, distance, surfel))
		{
			intersection = 1;
		}
	}

	for (size_t i = 0; i < data.numTriangles; i++)
	{
		Triangle triangle = data.triangles[i];

		if (triangle.intersectRay(ray, distance, surfel))
		{
			intersection = 1;
		}
	}


	if (intersection)
	{
		//intersection occured

		//find cos of incidence

		float cosI = dot(-ray.direction, surfel.normal);



		//--------------------------
		//		Direct Light
		//--------------------------
			//get material
		Material material = data.materials[surfel.materialIdx];

		vec3 directRadiance(0, 0, 0);

		//emit if material is emitter
		directRadiance += material.emmitance;

		//calculate direct light, iff ray is not inside of primitive
		bool inside = cosI < 0.0f;
		if (!inside) {
			directRadiance += shade(data, surfel, material, localState);
		}


		//--------------------------
		//		Scattering
		//--------------------------
		vec3 indirectRadiance = computeIndirectRadianceAndScatter(ray, surfel, material, cosI, distance, inside, localState);

		//save radiance
		ray.radiance0 += ray.radiance1 * directRadiance;
		ray.radiance1 *= indirectRadiance;
	}
	else
	{
		//ray completely missed
		ray.radiance0 += data.defaultColor * ray.radiance1;
		ray.active = false;
	}


	//update hiprand state
	data.curandStates[offset] = localState;

	//save the ray
	data.rays[offset] = ray;
}

__device__
vec3 computeIndirectRadianceAndScatter(Ray& ray, const SurfaceElement& surfel, const Material& material, float cosI, const float distance, const bool inside, hiprandState& localState)
{
	float r = hiprand_uniform(&localState);

	//--------------------------
	//		Diffuse
	//--------------------------
	if (material.diffAvg > 0.0f)
	{
		r -= material.diffAvg;

		if (r < 0.0f)
		{
			ray.origin = surfel.point + RAY_BUMP_EPSILON * surfel.normal;
			ray.direction = randomDirectionLambert(surfel.normal, localState);

			return material.diffuseColor;
		}

	}
	//--------------------------
	//		Reflected
	//--------------------------


	//both indexes of refraction, n1 / n2, and sin T squared
	float n1, n2, n, sinT2;

	//compute values that will be needed later
	computeSinT2AndRefractiveIndexes(material.indexOfRefraction, cosI, sinT2, n1, n2, n);

	//compute fresnel
	const float fresnelReflective = computeFresnelForReflectance(cosI, sinT2, n1, n2, n);

	if (material.specAvg > 0.0f)
	{
		//glossy reflection
		if (material.flags & MAT_FLAG_PURE_REFLECTION) {
			//pure reflectance: do not compute fresnel
			r -= material.specAvg;
		}
		else {
			//include fresnel
			r -= material.specAvg * fresnelReflective;
		}

		if (r < 0.0f)
		{
			//check for cook torrance
			if (material.flags & MAT_FLAG_COOK_TORRANCE)
			{
				//get a new importance-sampled normal according to the Beckmann ditribution
				vec3 beckmannNormal = randomDirectionBeckmann(surfel.normal, material.roughness, localState);

				//store the old incident direction
				vec3 incident = ray.direction;

				//reflect the ray according to the new normal
				reflRay(ray, surfel.point, beckmannNormal);


				//calculate the geometric term of Cook-Torrance, to account for shadowing and masking
				vec3 half = normalize(ray.direction - incident);

				float nh = abs(dot(surfel.normal, half));
				float nl = abs(dot(surfel.normal, ray.direction));
				float vh = abs(dot(incident, half));
				float nv = abs(cosI);

				float geometric = glm::min<float>(glm::min<float>(1, 2 * nh * nl / vh), 2 * nh * nv / vh);

				//note that the nl term in the denominator cancels out
				return material.specularColor * geometric / nv;
			}

			reflRay(ray, surfel, cosI);

			//TODO: Reference paper
			//glossy scattering
			if (material.specularExponent != INFINITY) {
				//use an importance sampled ray to determine which way the ray ought to travel
				ray.direction = randomDirectionPhong(ray.direction, material.specularExponent, localState);
			}
			//mirror reflectance otherwise.  Nothing to do

			return material.specularColor;
		}
	}
	//--------------------------
	//		Refracted
	//--------------------------
	if (material.refrAvg > 0.0f)
	{
		const float fresnelRefractive = 1.0f - fresnelReflective;

		//refraction
		r -= material.refrAvg * fresnelRefractive;

		if (r < 0.0f)
		{
			refrRay(ray, surfel, cosI, sinT2, n);

			//Apply Beer's law
			if (inside)
			{
				return vec3(
					expf(-distance * material.absorption.x),
					expf(-distance * material.absorption.y),
					expf(-distance * material.absorption.z));
			}
			//not inside, no need for absorption 
			return vec3(1, 1, 1);
		}
	}

	//ray was absorbed
	ray.active = false;
	return vec3(0, 0, 0);
}


__device__ 
Ray computeEyeRay(int x, int y, int dimX, int dimY, const Camera& camera, hiprandState& state)
{
	const float aspectRatio = float(dimY) / dimX;

	float jitteredX = x + hiprand_uniform(&state);
	float jitteredY = y + hiprand_uniform(&state);

	// Compute the side of a square at z = -1 (the far clipping plane) based on the 
	// horizontal left-edge-to-right-edge field of view

	//multiplying by negative 2 offsets the -.5 in the next step
	const float s = -2 * tan(camera.fieldOfView * 0.5f);

	// xPos / image.width() : map from 0 - 1 where the pixel is on the image
	const vec3 start = vec3(((jitteredX / dimX) - 0.5f) * s,
		1 * ((jitteredY / dimY) - 0.5f) * s * aspectRatio,
		1.0f)
		* camera.zNear;

	return Ray(camera.position, glm::normalize(camera.rotation * start));
}

__device__
vec3 shade(const RendererData& data, const SurfaceElement& surfel, const Material& material, hiprandState& state)
{
	vec3 w_i;
	float distance2;
	vec3 radiance = vec3(0, 0, 0);


	//loop through all point lights
	for (size_t i = 0; i < data.numPointLights; i++)
	{
		PointLight light = data.pointLights[i];

		if (lineOfSight(data, surfel.normal, surfel.point, light.position, w_i, distance2))
		{
			const vec3 L_i = light.power / float(4 * M_PI * distance2);

			float cosI = fmax(0.0f, dot(surfel.normal, w_i));

			radiance.x += cosI * L_i.r * material.diffuseColor.r * material.diffAvg * INVERSE_PI;
			radiance.y += cosI * L_i.g * material.diffuseColor.g * material.diffAvg * INVERSE_PI;
			radiance.z += cosI * L_i.b * material.diffuseColor.b * material.diffAvg * INVERSE_PI;
		}
	}

	//loop through all area lights
	for (size_t i = 0; i < data.numAreaLights; i++)
	{
		AreaLight light = data.areaLights[i];

		vec3 point = getAreaLightPoint(light, data.triangles, state);

		if (lineOfSight(data, surfel.normal, surfel.point, point, w_i, distance2))
		{
			const vec3 L_i = light.power / float(4 * M_PI * distance2);

			float cosI = fmax(0.0f, dot(surfel.normal, w_i));

			radiance.x += cosI * L_i.r * material.diffuseColor.r * material.diffAvg * INVERSE_PI;
			radiance.y += cosI * L_i.g * material.diffuseColor.g * material.diffAvg * INVERSE_PI;
			radiance.z += cosI * L_i.b * material.diffuseColor.b * material.diffAvg * INVERSE_PI;
		}
	}

	return radiance;
}

__device__
vec3 getAreaLightPoint(const AreaLight& light, Triangle* triangles, hiprandState& state) {
	//get a random point on a triangle
//	float u1 = hiprand_uniform(&state);
//	float u2 = hiprand_uniform(&state) * u1;
//	float weight0 = u1 - u2, weight1 = u2, weight2 = 1 - u1;

	float u1 = hiprand_uniform(&state);
	float u2 = hiprand_uniform(&state);
	float u3 = hiprand_uniform(&state);
	float inverseTotal = 1 / (u1 + u2 + u3);
	float weight0 = u1 * inverseTotal, weight1 = u2 * inverseTotal, weight2 = u3 * inverseTotal;
	
	//FIXME: This only works because the light is known to be rectangular - This needs to be expanded for more complex lights
	//get a random point on the light's triangles
	if (hiprand_uniform(&state) > .5f) {
		return triangles[light.triangleIdx].vertex0 * weight0 +
			triangles[light.triangleIdx].vertex1 * weight1 +
			triangles[light.triangleIdx].vertex2* weight2;
	}
	else {
		return triangles[light.triangleIdx + 1].vertex0 * weight0 +
			triangles[light.triangleIdx + 1].vertex1 * weight1 +
			triangles[light.triangleIdx + 1].vertex2* weight2;
	}
}

__device__
bool lineOfSight(const RendererData& data, const vec3& normal, const vec3& point0, const vec3& point1, vec3& w_i, float& distance2)
{
	const vec3 offset = point1 - point0;
	distance2 = dot(offset, offset);
	float distance = sqrt(distance2);

	w_i = offset / distance;

	const Ray losRay(point0 + (RAY_BUMP_EPSILON * normal), w_i);

	//shorten distance.
	distance -= 2 * RAY_BUMP_EPSILON;

	//loop through all primitives, seeing if any intersections occur
	SurfaceElement surfel;

	//TODO: More robust implementation
	for (size_t i = 0; i < data.numSpheres; i++)
	{
		Sphere sphere = data.spheres[i];

		if (sphere.intersectRay(losRay, distance, surfel))
		{
			return false;
		}
	}

	for (size_t i = 0; i < data.numTriangles; i++)
	{
		Triangle triangle = data.triangles[i];

		if (triangle.intersectRay(losRay, distance, surfel))
		{
			return false;
		}
	}

	return true;
}

__device__
float computeFresnelForReflectance(const float cosI, const float sinT2, const float n1, const float n2, const float n)
{
	//check for TIR
	if (sinT2 > 1.0f)
	{
		return 1.0;
	}

	const float cosT = sqrt(1.0f - sinT2);

	const float r_s = (n1 * cosI - n2 * cosT) / (n1 * cosI + n2 * cosT);
	const float r_p = (n2 * cosI - n1 * cosT) / (n2 * cosI + n1 * cosT);

	return (r_s * r_s + r_p * r_p) * 0.5f;
}

__device__
void computeSinT2AndRefractiveIndexes(const float refrIndex, float& cosI, float& sinT2, float& n1, float& n2, float& n)
{
	if (cosI > 0)
	{
		n2 = refrIndex;
		n1 = 1.0f;
	}
	else
	{
		//make sure cos I positive
		cosI = -cosI;

		n1 = refrIndex;
		n2 = 1.0f;
	}

	n = n1 / n2;

	sinT2 = n * n * (1.0f - cosI * cosI);
}

__device__
void reflRay(Ray& ray, const SurfaceElement& surfel, const float cosI)
{
	vec3 w_o = ray.direction - 2 * (-cosI) * surfel.normal;

	ray.origin = surfel.point + (surfel.normal * RAY_BUMP_EPSILON);
	ray.direction = w_o;
}

__device__
void reflRay(Ray& ray, const vec3& point, const vec3& normal)
{
	float cosI = abs(dot(ray.direction, normal));

	vec3 w_o = ray.direction - 2 * (-cosI) * normal;

	ray.origin = point + (normal * RAY_BUMP_EPSILON);
	ray.direction = w_o;
}

__device__
void refrRay(Ray& ray, const SurfaceElement& surfel, const float cosI, const float sinT2, const float n)
{
	//check for TIR
	if (sinT2 > 1.0f)
	{
		ray.active = false;
	}

	const float cosT = sqrt(1.0f - sinT2);

	vec3 w_o = normalize(n * ray.direction + (n * cosI - cosT) * surfel.normal);

	ray.origin = surfel.point + (w_o * RAY_BUMP_EPSILON);
	ray.direction = w_o;
}

__device__
vec3 randomDirectionLambert(vec3 const& normal, hiprandState& state)
{
	float theta = hiprand_uniform(&state) * 2 * M_PI;
	float s = hiprand_uniform(&state);
	float y = sqrt(s);
	float r = sqrt(1 - y * y);

	vec3 sample(r * cos(theta), y, r * sin(theta));
	quat rot = rotateVectorToVector(vec3(0, 1, 0), normal);

	return rot * sample;
}

__device__
vec3 randomDirectionPhong(const vec3& w_o, float exponent, hiprandState& state)
{
	float theta = hiprand_uniform(&state) * 2 * M_PI;
	float s = hiprand_uniform(&state);
	float y = pow(s, 1 / (exponent + 1));
	float r = sqrt(1 - y * y);

	vec3 sample(r * cos(theta), y, r * sin(theta));
	quat rot = rotateVectorToVector(vec3(0, 1, 0), w_o);

	return rot * sample;
}

vec3 randomDirectionBeckmann(vec3 const& normal, float roughness, hiprandState& state)
{
	float theta = atan(-roughness * roughness * log(1.0f - hiprand_uniform(&state)));
	float phi = hiprand_uniform(&state) * 2 * M_PI;

	float cosPhi = cosf(phi);
	float sinPhi = sinf(phi);
	float cosTheta = cosf(theta);
	float sinTheta = sinf(theta);
	//convert to cartesian coordinants from spherical
	vec3 m = vec3(sinTheta * cosPhi, cosTheta, sinTheta * sinPhi);

	quat rot = rotateVectorToVector(vec3(0, 1, 0), normal);

	return rot * m;
}

__device__
const quat rotateVectorToVector(const vec3& source, const vec3& target)
{
	vec3 axis = cross(source, target);
	quat rotation = quat(1.0f + dot(source, target), axis.x, axis.y, axis.z);
	return normalize(rotation);
}

void generateFrame(uchar4 *pixels, void* dataBlock, int ticks)
{
	ProgramData *data = (ProgramData *)dataBlock;

	//TODO: Implement a way to reset ticks when moving

	dim3 rayThreads(256);
	dim3 grids(DIM / 16, DIM / 16);
	dim3 threads(16, 16);

	//if necessary, clear the pixels and reset the number of ticks
	if (data->resetTicksThisFrame) {
		data->lastResetTick = ticks;

		clearPixels << <grids, threads >> >(pixels, data->totalPixelColors);

		data->resetTicksThisFrame = false;
	}

	//create the rays
	computeEyeRaysKernel << <grids, threads >> > (data->camera, data->renderData.rays, data->renderData.curandStates);

	int numRays = DIM * DIM;
	
	//fire n rays per pixel
	//number of iterations is only one if set to use ray tracing
	unsigned int numIterations = data->usePathTracer ? data->maxIterations : 1;
	for (unsigned int i = 0; i < numIterations && numRays != 0; i++) {
		dim3 rayGrids(numRays / 256);
		pathTraceKernel << < rayGrids, rayThreads >> >(pixels, data->renderData, i);

		if (i != numIterations - 1)
		{
			//Stream compaction
			thrust::device_ptr<Ray> dev_ray_ptr(data->renderData.rays);
			thrust::device_ptr<Ray> partitionRay = thrust::partition(dev_ray_ptr, dev_ray_ptr + numRays, ray_is_active());

			numRays = partitionRay - dev_ray_ptr;			
		}
	}

	//output number of rays cast thus far
	std::cout << "Rays per pixel: " << ticks - data->lastResetTick << "     \r";
	std::cout.flush();

	//write results to buffer
	writeToPixelsKernel << <grids, threads >> >(pixels, data->totalPixelColors, data->renderData.rays, ticks - data->lastResetTick);
}

int main(int argc, char *argv[])
{
	Scene scene;

	vec3 defaultColor(0, 0, 0);

	scene.build();

	PointLight *pointLights;
	AreaLight *areaLights;
	Ray* rays;
	Sphere *spheres;
	Triangle *triangles;
	Material *materials;
	hiprandState* curandStates;
	uint3* totalPixelColors;

	//initialize bitmap and data
	ProgramData *data = new ProgramData();
	GPUAnimBitmap bitmap(DIM, DIM, data);

	//allocate GPU pointers
	CUDA_ERROR_HANDLE(hipMalloc((void**)&pointLights,
		sizeof(PointLight)* scene.pointLightsVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&areaLights,
		sizeof(AreaLight)* scene.areaLightsVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&spheres,
		sizeof(Sphere)* scene.spheresVec.size()));
	
	CUDA_ERROR_HANDLE(hipMalloc((void**)&materials,
		sizeof(Material)* scene.materialsVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&triangles,
		sizeof(Triangle)* scene.trianglesVec.size()));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&rays,
		sizeof(Ray)* DIM * DIM));

	CUDA_ERROR_HANDLE(hipMalloc(&curandStates, 
		sizeof(hiprandState)* DIM *DIM));

	CUDA_ERROR_HANDLE(hipMalloc((void**)&totalPixelColors,
		sizeof(uint3)* DIM * DIM));

	//copy data to GPU
	CUDA_ERROR_HANDLE(hipMemcpy(spheres, scene.spheresVec.data(), sizeof(Sphere)* scene.spheresVec.size(), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(triangles, scene.trianglesVec.data(), sizeof(Triangle)* scene.trianglesVec.size(), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(pointLights, scene.pointLightsVec.data(), sizeof(PointLight)* scene.pointLightsVec.size(), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(areaLights, scene.areaLightsVec.data(), sizeof(AreaLight)* scene.areaLightsVec.size(), hipMemcpyHostToDevice));
	CUDA_ERROR_HANDLE(hipMemcpy(materials, scene.materialsVec.data(), sizeof(Material)* scene.materialsVec.size(), hipMemcpyHostToDevice));

	//put values in a data block
	data->camera = Camera();
	data->renderData.pointLights = pointLights;
	data->renderData.numPointLights = scene.pointLightsVec.size();
	data->renderData.areaLights = areaLights;
	data->renderData.numAreaLights = scene.areaLightsVec.size();
	data->renderData.spheres = spheres;
	data->renderData.numSpheres = scene.spheresVec.size();
	data->renderData.triangles = triangles;
	data->renderData.numTriangles = scene.trianglesVec.size();
	data->renderData.rays = rays;
	data->renderData.materials = materials;
	data->renderData.defaultColor = defaultColor;
	data->renderData.curandStates = curandStates;
	data->totalPixelColors = totalPixelColors;

	dim3 grids(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	curandSetupKernel << < grids, threads >> > (curandStates);



	bitmap.anim_and_exit((void(*)(uchar4*, void*, int))generateFrame, NULL, (void(*)(unsigned char, int, int))Key);

	//free
	CUDA_ERROR_HANDLE(hipFree(pointLights));
	CUDA_ERROR_HANDLE(hipFree(areaLights));
	CUDA_ERROR_HANDLE(hipFree(spheres));
	CUDA_ERROR_HANDLE(hipFree(triangles));
	CUDA_ERROR_HANDLE(hipFree(rays));
	CUDA_ERROR_HANDLE(hipFree(materials));
	CUDA_ERROR_HANDLE(hipFree(curandStates));
	CUDA_ERROR_HANDLE(hipFree(totalPixelColors));
	
	delete data;

	return 0;
}



// static method used for glut callbacks
void Key(unsigned char key, int x, int y) {

	GPUAnimBitmap*   bitmap = *(GPUAnimBitmap::get_bitmap_ptr());

	switch (key) {
		case 27:
		{
			   if (bitmap->animExit)
				   bitmap->animExit(bitmap->dataBlock);
			   bitmap->free_resources();
			   exit(0);
		}
		case 32:
		{
				   //space bar pressed: switch between path tracer and rudimentary ray tracer
				   ((ProgramData*)bitmap->dataBlock)->usePathTracer = !((ProgramData*)bitmap->dataBlock)->usePathTracer;
				   ((ProgramData*)bitmap->dataBlock)->resetTicksThisFrame = true;
		}
		case 48:
		{
				   //0 key pressed.  save image to file, with filename taken from current time

				   //get time
				   time_t     now = time(0);
				   struct tm  tstruct;
				   char       buf[160];
				   tstruct = *localtime(&now);

				   //get filename from time
				   strftime(buf, sizeof(buf), "renders/render%Y-%m-%d-%H%M%S.tga", &tstruct);
				   saveScreenshot(buf, DIM, DIM);
		}
	}

	if (moveCamera(((ProgramData*)bitmap->dataBlock)->camera, key))
	{
		((ProgramData*)bitmap->dataBlock)->resetTicksThisFrame = true;
	}
}


/// <summary>
/// Saves a screenshot.  Taken from http://www.flashbang.se/archives/155
/// </summary>
/// <param name="filename">The filename.</param>
/// <param name="x">The x dimensions.</param>
/// <param name="y">The y dimensions.</param>
void saveScreenshot(char filename[160], int x, int y)
{
	// get the image data
	long imageSize = x * y * 3;
	unsigned char *data = new unsigned char[imageSize];
	glReadPixels(0, 0, x, y, GL_BGR, GL_UNSIGNED_BYTE, data);// split x and y sizes into bytes
	int xa = x % 256;
	int xb = (x - xa) / 256; int ya = y % 256;
	int yb = (y - ya) / 256;//assemble the header
	unsigned char header[18] = { 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 0, 0, (char)xa, (char)xb, (char)ya, (char)yb, 24, 0 };
	// write header and data to file
	std::fstream File(filename, std::ios::out | std::ios::binary);
	File.write(reinterpret_cast<char *>(header), sizeof (char)* 18);
	File.write(reinterpret_cast<char *>(data), sizeof (char)*imageSize);
	File.close();

	delete[] data;
	data = NULL;
}


/// <summary>
/// Moves the camera.
/// </summary>
/// <param name="camera">The camera.</param>
/// <param name="key">The key that was pressed to move the camera.</param>
/// <returns>True if camera moved, false otherwise</returns>
bool moveCamera(Camera& camera, unsigned char key)
{
	float moveDist = .2f, rotateDist = 10.0f * M_PI / 180.0f;

	switch (key) {
	case 119:
		//forward (w)
		camera.position += camera.rotation * vec3(0, 0, -moveDist);
		return true;
	case 97:
		//left (a)
		camera.position += camera.rotation * vec3(-moveDist, 0, 0);
		return true;
	case 115:
		//backwards (s)
		camera.position += camera.rotation * vec3(0, 0, moveDist);
		return true;
	case 100:
		//right (d)
		camera.position += camera.rotation * vec3(moveDist, 0, 0);
		return true;
	case 113:
		//up (q)
		camera.position += camera.rotation * vec3(0, moveDist, 0);
		return true;
	case 101:
		//down (e)
		camera.position += camera.rotation * vec3(0, -moveDist, 0);
		return true;
	case 102:
		//rotate left (f)
		camera.rotation = glm::normalize(camera.rotation * quat(vec3(0, rotateDist, 0)));
		return true;
	case 104:
		//rotate right (h)
		camera.rotation = glm::normalize(camera.rotation * quat(vec3(0, -rotateDist, 0)));
		return true;
	case 103:
		//rotate down (g)
		camera.rotation = glm::normalize(camera.rotation * quat(vec3(-rotateDist, 0, 0)));
		return true;
	case 116:
		//rotate up (t)
		camera.rotation = glm::normalize(camera.rotation * quat(vec3(rotateDist, 0, 0)));
		return true;
	}

	return false;
}
